
/*
 * Erik Palmer
 * March 1, 2017
 *
 * This is stripped down version for public sharing. This code 
 * simulates an elastic dumbbells based on the 
 * Upper Convective Maxwell (UCM) model. 
 *
 * Global variables are used for important parameters, and 
 * computations are transferred between the CPU (host) and
 * GPU (device) as needed for optimal efficiency.
 *
 * Species switching dynamics have been removed so this 
 * will only simulate the evolution of "active" dumbbells.
 * 
 * To Compile:
 *      nvcc CUDA_FILENAME -lcurand -o EXECUTABLE_NAME 
 * 
 */




#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <errno.h>
#include <ctype.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
//#include <math_functions.h>


//Define Macros for Error handling

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
	printf("Error at %s:%d\n", __FILE__,__LINE__); \
	return EXIT_FAILURE; }} while(0)
#define CURAND_CALL(x) do { if((x)!= HIPRAND_STATUS_SUCCESS) { \
	printf("Error at %s:%d\n", __FILE__, __LINE__); \
	return EXIT_FAILURE; }} while(0)

//Define Macro for Histogram debugging
#define PRINT_VAR(x) printf("" #x  "\n ")

//Debugging Macros
#define PRINT_VAR_FLOAT_VALUE(x) printf("" #x "=%f\n", x)
#define PRINT_VAR_INT_VALUE(x) printf("" #x "=%d\n", x) 
//* Also useful: printf("DEBUG LINE %d\n", __LINE__);


//___velocity field on-off matrix ____ 
// note that this matrix is multiplied by the inputted flowrate value
#define U11 0.0
#define U12 0.0 
#define U21 1.0 
#define U22 0.0 
//``````````````````````````````


//___Simulation Constants from paper____
#define INIT_ACT_TO_DNG_RATIO 0.5 
#define LITTLE_D 0.03 				//Default 0.03
#define TAO_FUND 5e-6 				//Default 5e-6
#define ZEE 10.0 	 			//Default 10.0
#define ALPHA_ZERO 0.83				//Default 0.83
#define ALPHA_ONE 0.17				//Default 0.17
#define U_ZERO 14.0      			//Default 14.0
#define D_FREE 12.0				//Default 12.0

//``````````````````````````````



//____Define Global Variables________
//For GPU
__device__ double devStepSizeMicro;
__device__ unsigned int devTimeStepsMicro;
__device__ double devFlowRate;
__device__ double devMaxSpringLength;
__device__ double devFreq;

//For CPU
static long hostNumberOfParticles = 0;
static double hostStepSizeMicro = 0;
static long hostTimeStepsMicro = 0;
static long hostTimeStepsMacro = 0;
static double hostFlowRate;
static double hostMaxSpringLength = 0;
static double hostFreq;
//``````````````````````````````````


/*
 * Function: ParseInput
 * Sorts and examines command line input for inappropriate data
 */
int ParseInput(int argc, char *argv[]){

	int i;

	if (argc != 7){	
		printf("ERROR: Incorrect number of input arguments\n");
		printf("Format: ./Maxwell [number of particles] [micro step size]");
		printf(" [time steps micro] [time steps macro] [flow rate]"); //Max Spring Length Removed
		printf(" [SAOS frequency]\n"); 
		return EXIT_FAILURE; 
	}

	char *argvCopy;



printf("The running program is %s\n", argv[0]);

	for (i=1; i<argc; i++){
		argvCopy = argv[i];

		for (; *argv[i]!='\0'; argv[i]++){
			if (*argv[i]=='.') continue; //skip decimals
			if (isdigit(*argv[i])==0){
				printf("%s is not a number\n", argv[i]);
				return EXIT_FAILURE;
			}
		}
		argv[i] = argvCopy;
	}
	
	errno = 0; 

	hostNumberOfParticles = strtol(argv[1], NULL, 10);
	hostStepSizeMicro = strtod(argv[2], NULL);
	hostTimeStepsMicro = strtol(argv[3], NULL, 10);
	hostTimeStepsMacro = strtol(argv[4], NULL, 10); 	
	hostFlowRate = strtod(argv[5], NULL);
	//hostMaxSpringLength = strtod(argv[6], NULL);  //turned off for UCM Maxwell 
	hostFreq = strtod(argv[6], NULL);
	
	if (hostNumberOfParticles==0){
		printf("Unable to convert %s to positive integer\n", argv[1]);
		return EXIT_FAILURE;
	}
	if (hostTimeStepsMicro==0){
		printf("Unable to convert %s to positive integer\n", argv[3]);
		return EXIT_FAILURE;
	}
	if (hostTimeStepsMacro==0){
		printf("Unable to convert %s to positive integer\n", argv[4]);
		return EXIT_FAILURE;
	}
	if (hostStepSizeMicro==0){
		printf("Unable to convert %s to double\n", argv[2]);
		return EXIT_FAILURE;
	}
	//commented out to allow zero flow rate
	/*
	if (hostFlowRate==0.0){
		printf("Unable to convert %s to positive double\n", argv[5]);
		return EXIT_FAILURE;
	}
	*/

	//commented out for UCM Maxwell
	/*
	if (hostMaxSpringLength == 0){
		printf("Unable to convert %s to positive double\n", argv[6]);
		return EXIT_FAILURE;
	}
	*/
	if (hostFreq == 0){
		printf("Unable to convert %s to positive double\n", argv[6]);
		return EXIT_FAILURE;
	}

	if (errno == ERANGE){
		printf("%s\n", strerror(errno));
		return EXIT_FAILURE;	
	}


	return 0;
}

/*
 * Function PrintSimInfo
 * Prints to terminal information about the current simulation
 */

void PrintSimInfo(){

	// ___ Calculate and output program parameters _____

	printf("___________Running Steady State UCM Maxwell Simulation_________________\n");
	printf("|| Number of Particles: %d\n", hostNumberOfParticles);
	printf("|| Total Time: %g \n", hostTimeStepsMicro * hostStepSizeMicro * hostTimeStepsMacro);
	printf("|| Flow Rate: %g \n", hostFlowRate);
	printf("|| Macro -- Steps: %d, Step Size: %g\n", hostTimeStepsMacro, hostTimeStepsMicro * hostStepSizeMicro);
	printf("|| Micro -- Steps: %d, Step Size: %1.12g\n", hostTimeStepsMicro, hostStepSizeMicro);
	printf("|| Maximum Spring Length: %g\n", hostMaxSpringLength );
	printf("|| SAOS Frequency: %g\n", hostFreq );
	printf(" - - - - - - - - - - - - - - - - - - - - - - - \n");
	
	//``````````````````````````````````````````````````
}


/*
 * Function OutputToFile
 * Writes header containing information about the simulation
 * and contents of three vectors to csv file
 */

void OutputToFile (double XX[], double XY[], double YY[], double time_spent, int count){
	
	
	FILE *OutputFile;
	char OutputFileName[] = "MaxwellSSimData";

	sprintf(OutputFileName, "%s.csv", OutputFileName); //<---Filename

	OutputFile = fopen(OutputFileName, "w");

	if (OutputFile == NULL){
		fprintf(stderr, "Couldn't open output file: %s!\n", OutputFileName);
		exit(1);	
	}

	// ____ Header for textfile _______________________
	

	//Description 	

	fprintf(OutputFile,"**********************************************************************\n");
	fprintf(OutputFile,"*     Simulation For UCM Maxwell                                     *\n"); 
	fprintf(OutputFile,"*                                                                    *\n"); 
	fprintf(OutputFile,"*                                                                    *\n"); 
	fprintf(OutputFile,"*                                                                    *\n"); 
	fprintf(OutputFile,"*                                                                    *\n"); 
	fprintf(OutputFile,"*                                                                    *\n"); 
	fprintf(OutputFile,"**********************************************************************\n");
	
	fprintf(OutputFile,"TotalTime: %3.12g\n", hostTimeStepsMicro * hostStepSizeMicro * hostTimeStepsMacro);
	fprintf(OutputFile,"FlowRate: %g\n", hostFlowRate);
	fprintf(OutputFile,"MacroSteps: %ld\n", hostTimeStepsMacro);
	fprintf(OutputFile,"MacroStepSize: %3.12g\n", hostTimeStepsMicro * hostStepSizeMicro);
	fprintf(OutputFile,"MicroSteps: %ld\n", hostTimeStepsMicro);
	fprintf(OutputFile,"StepSize: %2.12g\n", hostStepSizeMicro);
	fprintf(OutputFile,"NumberOfParticles: %ld\n", hostNumberOfParticles);
	fprintf(OutputFile,"Runtime: %g\n", time_spent);
	fprintf(OutputFile,"MaxSpringLength: %g\n", hostMaxSpringLength);
	fprintf(OutputFile,"SAOSFrequency: %g\n", hostFreq);
	fprintf(OutputFile,"Initial-Active-to-Dangling-Ratio: %g\n", INIT_ACT_TO_DNG_RATIO);
	fprintf(OutputFile,"Potential-well-distance(d): %g\n", LITTLE_D );
	fprintf(OutputFile,"Tao_Fundamental: %g\n", TAO_FUND);
	fprintf(OutputFile,"Z: %g\n", ZEE);
	fprintf(OutputFile,"Alpha_Zero: %g\n", ALPHA_ZERO);
	fprintf(OutputFile,"Alpha_One: %g\n", ALPHA_ONE);
	fprintf(OutputFile,"U_Zero: %g\n", U_ZERO);
	fprintf(OutputFile,"D_Free: %g\n", D_FREE);

	//`````````````````````````````````````````````


	//____ print ensemble average at each macro time step ______ 
	fprintf(OutputFile," - - - - - - - - - - - - - - - - - - - - - - - \n");
	fprintf(OutputFile,"||   XX    ||   XY    ||   YY   ||\n");
	fprintf(OutputFile," - - - - - - - - - - - - - - - - - - - - - - - \n");

	int k;
	for (k=0; k<count; k++){
		fprintf(OutputFile,"% 2.16g,"  , XX[k]);
		fprintf(OutputFile," % 2.16g," , XY[k]);
		fprintf(OutputFile," % 2.16g\n", YY[k]);

	
	}
	//```````````````````````````````````````````````````````

	fclose(OutputFile); 

}


/*
 * Function: 
 * GPU Function
 * Calculates the change of state probability of an active dumbbell 
 * given the spring length
 * Tao must be computed each time: See paper, use equations 10 AND 11.
 */

__device__ double ActiveToDanglingProb (double SpringLen){
        double Tao_zero = TAO_FUND * exp ( U_ZERO ); //Equation (11)  //INEFFICIENT - this computation can be moved out of loop	
	
	//__ HOOK Sim__dimensional__
	double Tao = Tao_zero * exp ( - ( LITTLE_D * LITTLE_D * SpringLen * SpringLen) / U_ZERO ); //Equation (10)
	//````````````````

	return 1.0 - exp( -2.0 * devStepSizeMicro / Tao ); //Equation (13)


}


/*
 * Function: 
 * GPU Function
 * Calculates the change of state probability for a dangling dumbbell. 
 */

__device__ double DanglingToActiveProb (double SpringLen) {

	//__ Hook Sim _____
	return 1.0 - exp( - (ALPHA_ZERO + ALPHA_ONE * SpringLen) * devStepSizeMicro); //Equation (14) 
	//`````````````````

}


/*
 * Function: EvolveActive
 * GPU Function
 * Evolve Active Dumbbell for one micro step on GPU
 */ 

__device__ void EvolveActive (double *SpringLenX, double *SpringLenY, double randx, double randy, double *AvgSpringLifes, double *SimTime,
				double totaltime){
  
	double SpringLenXStep, SpringLenYStep;
	

	double drag_coeff_active = 0.5;  // set to 0.5 for comparison with analytic UCM result
	

	//_____  Non-Dim Evo-Equations  
	
	SpringLenXStep = *SpringLenX //;
		+ (U11 * *SpringLenX + U21 * devFreq * cos(devFreq * *SimTime) * *SpringLenY) * devStepSizeMicro * devFlowRate
		- drag_coeff_active * *SpringLenX * devStepSizeMicro 
		+ sqrt( devStepSizeMicro ) * randx; 
		//````````````````````````````


	SpringLenYStep = *SpringLenY //
		+ (U12 * *SpringLenX + U22 * *SpringLenY) * devStepSizeMicro * devFlowRate
		- drag_coeff_active * *SpringLenY * devStepSizeMicro
		+ sqrt( devStepSizeMicro ) * randy; 
		//```````````````````````````
	
	//``````````````````````````````````````

	*SpringLenX = SpringLenXStep;
	*SpringLenY = SpringLenYStep;

}


/*
 * Function: EvolveDangling
 * GPU Function
 * Evolve Dangling Dumbbell for one micro step on GPU
 */

__device__ void EvolveDangling(double *SpringLenX, double *SpringLenY, double randx, double randy, double *AvgSpringLifes, double *SimTime,
				double totaltime){
  
	double SpringLenXStep, SpringLenYStep;
	
	double drag_coeff_dangle = 0.5; //For comparison with UCM


	//_____ Old Non-Dim Evo-Equations: Hook dumbbells ______
		
	SpringLenXStep = *SpringLenX
		+ (U11 * *SpringLenX + U21 * devFreq * cos(devFreq * *SimTime) * *SpringLenY) * devStepSizeMicro * devFlowRate
		- drag_coeff_dangle * *SpringLenX * devStepSizeMicro
		+ sqrt( drag_coeff_dangle * devStepSizeMicro ) * randx;		
		
	SpringLenYStep = *SpringLenY
		+ (U12 * *SpringLenX + U22 * *SpringLenY) * devStepSizeMicro * devFlowRate
		- drag_coeff_dangle * *SpringLenY * devStepSizeMicro
		+ sqrt( drag_coeff_dangle * devStepSizeMicro ) * randy;
		
	//```````````````````````````````````````````

	*SpringLenX = SpringLenXStep;
	*SpringLenY = SpringLenYStep;

}


/* 
 * Function: Micro_Steps
 * Loops through the Micro loop of the SDE
 */

__global__ void Micro_Steps(	double *SpringLenX, double *SpringLenY, int *SpeciesType,
				hiprandState *states, hiprandState *ProbStates,
				double *AvgSpringLifes, double *SimTime, double totaltime){
		
	int i = threadIdx.x + blockIdx.x * blockDim.x;	
	

	//___Device API for Random Number Generation____
	//copy state to local state for efficiency
	hiprandState localState = states[i];
	hiprandState localProbState = ProbStates[i];

	int j;	
	

	//TODO: Move node value calculation here, since it only changes once each time this function is called. 

	double2 RandNorm;
    //double RandUniform;
	//double SpringLen;

	for(j=0; j < devTimeStepsMicro; j++){
	
		//generate new random number each time
		RandNorm = hiprand_normal2_double(&localState);
		//RandUniform = curand_uniform_double(&localProbState);		//Disabled because species switching turned off

		//Calculate Spring Length
		//SpringLen = sqrt(SpringLenX[i] * SpringLenX[i] + SpringLenY[i] * SpringLenY[i]); //Disabled b/c species switching turned off

		

		//_____Evolve Dumbbells According to their species_______
		if (SpeciesType[i]==0){   //if active type
			EvolveActive(&SpringLenX[i], &SpringLenY[i], RandNorm.x, RandNorm.y, AvgSpringLifes, &SimTime[i], totaltime);
		}
		else if (SpeciesType[i]==1){ //if dangling type
			EvolveDangling(&SpringLenX[i], &SpringLenY[i], RandNorm.x, RandNorm.y, AvgSpringLifes, SimTime, totaltime);
		}
		//`````````````````````````````````````````````````````````

		SimTime[i] += devStepSizeMicro; 

	}

	//copy random number generator state back
	states[i] = localState;
	ProbStates[i] = localProbState;

}


/* 
 * Function: RandomGenInit
 * Initialize the random number generator on each of the threads
 * Gives each thread a different seed form *SeedList vector
 */

__global__ void RandomGenInit(unsigned int *SeedList, hiprandState *states){

	int tid  = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(SeedList[tid], tid, 0, &states[tid]);
}

__global__ void PrintSpringLengths ( double *SpringLenX, double *SpringLenY) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	printf(" x:% f y:% f \n", SpringLenX[tid], SpringLenY[tid]);
}




/*
 * Function: RndNorm
 * CPU Function to transform uniform random variable [0,1] to normal random variable
 * with mean 0 and Variance defined in the function
 */

__host__ double RndNorm (void)
{
	double Variance = 1; 
	 
	static int HasSpareRandomNum = 0;
	static double SpareRandomNum;

	if(HasSpareRandomNum == 1){
	        HasSpareRandomNum = 0;
		return Variance * SpareRandomNum;  	       
	}

	HasSpareRandomNum = 1;
	
	static double u,v,s;

	do{
		u = (  rand() / ((double) RAND_MAX)) * 2 - 1;
		v = (  rand() / ((double) RAND_MAX)) * 2 - 1;
		s = u * u + v * v;	
	} while (s >= 1 || s == 0); 

	s = sqrt (-2.0 * log(s) / s);
	
        SpareRandomNum = v * s;   //Save spare random number for next function call

	return Variance * u * s;	
}



__host__ void OutputRatio (int Active, int Dangling){
	double ActivePercent =  (double)Active / hostNumberOfParticles; 
	double DanglingPercent = (double)Dangling / hostNumberOfParticles;
	printf("Active %f Dangling: %f\n", ActivePercent, DanglingPercent); 
}



__host__ double AvgSpringLife ( double *SpringLenX, double *SpringLenY, int *SpeciesType){
	int j;
        double Tao_zero = TAO_FUND * exp ( U_ZERO );	
	double Total = 0.0; 
	double SpringLen;
	int ActiveCount = 0;
	for (j=0; j<hostNumberOfParticles; j++){

		if (SpeciesType[j] == 0){	//If active type
		ActiveCount++;
		SpringLen = sqrt( SpringLenX[j] * SpringLenX[j] + SpringLenY[j] * SpringLenY[j]); 
		
		//__Hookean Springs__
		Total += Tao_zero * exp (- LITTLE_D * LITTLE_D * SpringLen * SpringLen / U_ZERO ); 	
		//``````````````````

		}
		
	}
	
	return Total / (double) ActiveCount;

} 





int main(int argc, char *argv[]){
	
	//_____Record Program Run Time
	clock_t begin, end, end2;
	begin = clock();
	double time_spent, time_spent2;
	//````````````````````````````````



	// ____  Read Command Line Arguments _____
		
	if (ParseInput(argc, argv)==EXIT_FAILURE){
		exit(2);
	}
	//`````````````````````````````````````


	PrintSimInfo(); //Output Simulation Variables to Terminal 



	//___ Set Global Variable Values _______
	hipMemcpyToSymbol(HIP_SYMBOL(devStepSizeMicro), &hostStepSizeMicro, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(devTimeStepsMicro), &hostTimeStepsMicro, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(devFlowRate), &hostFlowRate, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(devMaxSpringLength), &hostMaxSpringLength, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(devFreq), &hostFreq, sizeof(double));
	//```````````````````````````````````````


	//____define block and thread structure______
	dim3 block;
	
	if (hostNumberOfParticles < 32){
		block.x = hostNumberOfParticles;
		block.y = 1;	
	}
	else {
		block.x=512;
		block.y = 1;
	}

	dim3 grid ((hostNumberOfParticles + block.x -1) / block.x,1);
	//`````````````````````````````````````



	//__Variables for random number generation on GPU kernels
	hiprandState *states = NULL;
	hiprandState *ProbStates = NULL;
	//``````````````````````````````````

	//____allocate memory on GPU for random number generator states______
	CUDA_CALL(hipMalloc((void **)&states, sizeof(hiprandState) * hostNumberOfParticles ));
	CUDA_CALL(hipMalloc((void **)&ProbStates, sizeof(hiprandState) * hostNumberOfParticles ));
	//`````````````````````````````````````````````````````````````````
	
	//__create vectors of seeds_____
	unsigned int *hostSeeds, *devSeeds;

	unsigned int *hostProbSeeds, *devProbSeeds;


	hostSeeds = (unsigned int *)malloc(hostNumberOfParticles*sizeof(unsigned int));	
	hostProbSeeds = (unsigned int *)malloc(hostNumberOfParticles*sizeof(unsigned int));


	CUDA_CALL(hipMalloc((void **)&devSeeds, sizeof(unsigned int) * hostNumberOfParticles));
	CUDA_CALL(hipMalloc((void **)&devProbSeeds, sizeof(unsigned int) * hostNumberOfParticles));
	
	srand(time(NULL));
	
	int i;
	for (i=0; i<hostNumberOfParticles; i++){
		hostSeeds[i] = rand();
		hostProbSeeds[i] = rand();
	}
	//````````````````````````````
	



	CUDA_CALL(hipMemcpy(devSeeds, hostSeeds, sizeof(unsigned int) * hostNumberOfParticles, hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(devProbSeeds, hostProbSeeds, sizeof(unsigned int) * hostNumberOfParticles, hipMemcpyHostToDevice));
	

	//___initialize kernel random number generator on GPU threads____
	RandomGenInit<<< grid, block >>>(devSeeds, states);
	CUDA_CALL( hipPeekAtLastError() ); 
	CUDA_CALL( hipDeviceSynchronize() );
	RandomGenInit<<< grid, block >>>(devProbSeeds, ProbStates);
	CUDA_CALL( hipPeekAtLastError() ); 
	CUDA_CALL( hipDeviceSynchronize() );
	//````````````````````````````````````





	//____Spring Length variables____
	double *devSpringLenX, *devSpringLenY; 
	double *hostSpringLenX, *hostSpringLenY;
	//`````````````````````````````````

	//___Dumbbell Species Type Variable___
	int *devSpeciesType; 
	int *hostSpeciesType;
	//``````````````````````````````````

	//_______allocate memory on CPU 
	hostSpringLenX = (double*)malloc(hostNumberOfParticles*sizeof(double));
	hostSpringLenY = (double*)malloc(hostNumberOfParticles*sizeof(double));
	hostSpeciesType = (int*)malloc(hostNumberOfParticles*sizeof(int));
	//`````````````````````````

	//_____allocate memory on GPU for spring length
	CUDA_CALL(hipMalloc((double**)&devSpringLenX, hostNumberOfParticles*sizeof(double)));
	CUDA_CALL(hipMalloc((double**)&devSpringLenY , hostNumberOfParticles*sizeof(double))); 
	CUDA_CALL(hipMalloc((int**)&devSpeciesType, hostNumberOfParticles*sizeof(int)));
	//`````````````````````````````````````


	//___Simulation Time____
	//Variables for tracking time t throughout simulation
	double *devSimTime, *hostSimTime;

	hostSimTime = (double *)malloc(hostNumberOfParticles*sizeof(double));
	CUDA_CALL(hipMalloc((double**)&devSimTime,hostNumberOfParticles*sizeof(double)));
	//````````````````````````````````````````````



	//___ Set initial Spring Lengths to Normal Distribution
	
	int l; 

	for (l=0; l < hostNumberOfParticles; l++){

		hostSimTime[l] = 0.0;



		//___ Set initial length randomly__
		hostSpringLenX[l] = RndNorm();	//Starting from this appears to speed up
		hostSpringLenY[l] = RndNorm();	// steady state for SAOS	 
		//`````````````````````````````````

		
		//___set initial species type__
		
		hostSpeciesType[l] = 0; //Make all dumbbells active initially

		//`````````````````````````````

	}
	//``````````````````````````````````````````````````


	//____Copy spring lengths to Gpu device
	CUDA_CALL(hipMemcpy(devSpringLenX, hostSpringLenX, hostNumberOfParticles*sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(devSpringLenY, hostSpringLenY, hostNumberOfParticles*sizeof(double), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(devSpeciesType, hostSpeciesType, hostNumberOfParticles*sizeof(int), hipMemcpyHostToDevice));
	//````````````````````````````````````````````````	

	CUDA_CALL(hipMemcpy(devSimTime, hostSimTime, hostNumberOfParticles*sizeof(double), hipMemcpyHostToDevice));



	//PrintSpringLengths<<< grid, block >>>(devSpringLenX, devSpringLenY); //print lengths to verify created correctly




	//___ initialize variables to calculate and store ensemble average 
	double *Spring_AvgLen_XX;
	double *Spring_AvgLen_XY;
	double *Spring_AvgLen_YY;

	Spring_AvgLen_XX = (double*)malloc((hostTimeStepsMacro+1)*sizeof(double));
	Spring_AvgLen_XY = (double*)malloc((hostTimeStepsMacro+1)*sizeof(double));
	Spring_AvgLen_YY = (double*)malloc((hostTimeStepsMacro+1)*sizeof(double));

	int k;

	double EnsembleAverageXX_Active = 0.0;
	double EnsembleAverageXY_Active = 0.0;
	double EnsembleAverageYY_Active = 0.0;
	
	double EnsembleAverageXX_Dangling = 0.0;
	double EnsembleAverageXY_Dangling = 0.0;
	double EnsembleAverageYY_Dangling = 0.0;


	int j;
	//````````````````````````````````````````````````````````



	int NumberOfActive = 0;
	int NumberOfDangling = 0;

	//_____calculate ensemble average at time = 0 
	for (j=0; j<hostNumberOfParticles; j++){

		if (hostSpeciesType[j]==0){ //if dumbbell is Active type
			NumberOfActive++;
						
			//___Hookean Springs____
			
			EnsembleAverageXX_Active += - hostSpringLenX[j] * hostSpringLenX[j];
			EnsembleAverageXY_Active += - hostSpringLenX[j] * hostSpringLenY[j];
			EnsembleAverageYY_Active += - hostSpringLenY[j] * hostSpringLenY[j];
			
			//```````````````````
			

		} else if (hostSpeciesType[j]==1){ //if dumbbell is Dangling type
			NumberOfDangling++;
						
			//___Hookean Springs____
			
			EnsembleAverageXX_Dangling += -hostSpringLenX[j] * hostSpringLenX[j];
			EnsembleAverageXY_Dangling += -hostSpringLenX[j] * hostSpringLenY[j];
			EnsembleAverageYY_Dangling += -hostSpringLenY[j] * hostSpringLenY[j];
			
			//```````````````````
			
		} else {
			printf("Error1: Unable to Classify Species Type\n");
		}
	}




	if (NumberOfActive == 0){
		
		Spring_AvgLen_XX[0] = EnsembleAverageXX_Dangling / (double)NumberOfDangling;
		Spring_AvgLen_XY[0] = EnsembleAverageXY_Dangling / (double)NumberOfDangling;
		Spring_AvgLen_YY[0] = EnsembleAverageYY_Dangling / (double)NumberOfDangling;

	} else if ( NumberOfDangling == 0){
	
		Spring_AvgLen_XX[0] = EnsembleAverageXX_Active / (double)NumberOfActive;
		Spring_AvgLen_XY[0] = EnsembleAverageXY_Active / (double)NumberOfActive;
		Spring_AvgLen_YY[0] = EnsembleAverageYY_Active / (double)NumberOfActive;
	
	} else {

	Spring_AvgLen_XX[0] = EnsembleAverageXX_Active / (double)NumberOfActive + EnsembleAverageXX_Dangling / (double)NumberOfDangling;
	Spring_AvgLen_XY[0] = EnsembleAverageXY_Active / (double)NumberOfActive + EnsembleAverageXY_Dangling / (double)NumberOfDangling;
	Spring_AvgLen_YY[0] = EnsembleAverageYY_Active / (double)NumberOfActive + EnsembleAverageYY_Dangling / (double)NumberOfDangling;
	
	}



	//``````````````````````````````````
	
	//____To Calculate Average Length of all Active Dumbbells___
	double *hostAverageSpringLife, *devAverageSpringLife;
	
	hostAverageSpringLife = (double *)malloc(sizeof(double));
	CUDA_CALL(hipMalloc((double**)&devAverageSpringLife,sizeof(double))); 
	//```````````````````````````````````````````````````````

	double totaltime = hostStepSizeMicro * hostTimeStepsMicro * hostTimeStepsMacro;





	//_____ Main simulation loop (Macro Time) ____


	for (k=1; k<=hostTimeStepsMacro; k++){



		//Calculate Average Length of all Active dumbbells		
		*hostAverageSpringLife = AvgSpringLife(hostSpringLenX, hostSpringLenY, hostSpeciesType); 
		CUDA_CALL(hipMemcpy(devAverageSpringLife,hostAverageSpringLife,sizeof(double),hipMemcpyHostToDevice));
		
		//Call function to perform computations on GPU
		Micro_Steps<<<grid,block>>>(devSpringLenX,devSpringLenY,devSpeciesType,states,ProbStates,devAverageSpringLife,devSimTime,totaltime);

		//read result from gpu(device) back to cpu(host)
		CUDA_CALL(hipMemcpy(hostSpringLenX, devSpringLenX, hostNumberOfParticles*sizeof(double), hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(hostSpringLenY, devSpringLenY, hostNumberOfParticles*sizeof(double), hipMemcpyDeviceToHost));
		CUDA_CALL(hipMemcpy(hostSpeciesType, devSpeciesType, hostNumberOfParticles*sizeof(int), hipMemcpyDeviceToHost));
	
		//read sim time back from gpu(device) back to cpu(host)
		CUDA_CALL(hipMemcpy(hostSimTime, devSimTime, sizeof(double), hipMemcpyDeviceToHost)); 
		


		NumberOfActive = 0;
		NumberOfDangling = 0;

		EnsembleAverageXX_Active = 0;
		EnsembleAverageXY_Active = 0;
		EnsembleAverageYY_Active = 0;
		EnsembleAverageXX_Dangling = 0;
		EnsembleAverageXY_Dangling = 0;
		EnsembleAverageYY_Dangling = 0;



		//____ Calculate Ensemble Averages ______
		for (j=0; j<hostNumberOfParticles; j++){


			if (hostSpeciesType[j]==0){ //if dumbbell is Active type
				NumberOfActive++;
				
				//___Hookean Springs____
				
				EnsembleAverageXX_Active += -hostSpringLenX[j] * hostSpringLenX[j];
				EnsembleAverageXY_Active += -hostSpringLenX[j] * hostSpringLenY[j];
				EnsembleAverageYY_Active += -hostSpringLenY[j] * hostSpringLenY[j];
				
				//```````````````````
				

			} else if (hostSpeciesType[j]==1){ //if dumbbell is Dangling type
				NumberOfDangling++;
				//____Hookean Springs_____
					
				EnsembleAverageXX_Dangling += -hostSpringLenX[j] * hostSpringLenX[j];
				EnsembleAverageXY_Dangling += -hostSpringLenX[j] * hostSpringLenY[j];
				EnsembleAverageYY_Dangling += -hostSpringLenY[j] * hostSpringLenY[j];
				
				//````````````````````````````	

			} else {
				printf("Error2: Unable to Classify Species Type of Dumbbell[%d] with Type: %d \n", j, hostSpeciesType[j]);
				exit(4);
			}

		}
		

		if (NumberOfActive == 0){
			
			Spring_AvgLen_XX[k] = EnsembleAverageXX_Dangling / (double)NumberOfDangling;
			Spring_AvgLen_XY[k] = EnsembleAverageXY_Dangling / (double)NumberOfDangling;
			Spring_AvgLen_YY[k] = EnsembleAverageYY_Dangling / (double)NumberOfDangling;

		} else if ( NumberOfDangling == 0){
		
			Spring_AvgLen_XX[k] = EnsembleAverageXX_Active / (double)NumberOfActive;
			Spring_AvgLen_XY[k] = EnsembleAverageXY_Active / (double)NumberOfActive;
			Spring_AvgLen_YY[k] = EnsembleAverageYY_Active / (double)NumberOfActive;
		
		} else {
 
		Spring_AvgLen_XX[k] = EnsembleAverageXX_Active / (double)NumberOfActive + EnsembleAverageXX_Dangling / (double)NumberOfDangling;
		Spring_AvgLen_XY[k] = EnsembleAverageXY_Active / (double)NumberOfActive + EnsembleAverageXY_Dangling / (double)NumberOfDangling;
		Spring_AvgLen_YY[k] = EnsembleAverageYY_Active / (double)NumberOfActive + EnsembleAverageYY_Dangling / (double)NumberOfDangling;
		
		}




	}
	//``````````````End Macro loop``````````````



	// __ stop computational clock ____	
	end = clock();
	time_spent = double(end-begin)/ CLOCKS_PER_SEC;
	//````````````````````````````````


	//___Write Values to .csv file
	OutputToFile(Spring_AvgLen_XX, Spring_AvgLen_XY, Spring_AvgLen_YY, time_spent, k);
	//````````````````````````````

	
	OutputRatio(NumberOfActive,NumberOfDangling);

	//___ clean up memory ____

	free(hostSimTime);
	CUDA_CALL(hipFree(devSimTime)); 

	free(hostAverageSpringLife);
	CUDA_CALL(hipFree(devAverageSpringLife));

	
	free(hostSeeds);
	CUDA_CALL(hipFree(devSeeds));
	CUDA_CALL(hipFree(states));
	
	free(hostProbSeeds);
	CUDA_CALL(hipFree(devProbSeeds));
	CUDA_CALL(hipFree(ProbStates));
	
	free(hostSpringLenX);
	free(hostSpringLenY);
	CUDA_CALL(hipFree(devSpringLenX));
	CUDA_CALL(hipFree(devSpringLenY));



	free(Spring_AvgLen_XX);
	free(Spring_AvgLen_XY);
	free(Spring_AvgLen_YY);

	
	//```````````````````````

	hipDeviceReset();


	// __ stop computational clock ____	
	end2 = clock();
	time_spent2 = double(end2-begin)/ CLOCKS_PER_SEC;
	printf("Runtime: %f\n\n", time_spent2);
	//````````````````````````````````
	 

	return EXIT_SUCCESS; 

}


